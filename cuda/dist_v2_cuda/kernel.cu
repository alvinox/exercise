#include "hip/hip_runtime.h"
#include "kernel.h"

#define TPB 32

__device__ float distance(float x1, float x2) {
  return sqrt((x2-x1)*(x2-x1));
}

__global__ void distanceKernel(float* d_out, float* d_in, float ref) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const float x = d_in[i];
  d_out[i] = distance(x, ref);
  printf("i = %2d: dist from %f to %f is %f.\n", i, ref, x, d_out[i]);
}

void distanceArray(float* out, float* in, float ref, int n) {
  float* d_in = NULL;
  float* d_out = NULL;

  hipMalloc(&d_in, n * sizeof(float));
  hipMalloc(&d_out, n * sizeof(float));

  hipMemcpy(d_in, in, n * sizeof(float), hipMemcpyHostToDevice);

  distanceKernel<<<n/TPB, TPB>>>(d_out, d_in, ref);

  hipMemcpy(out, d_out, n * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_in);
  hipFree(d_out);
}
