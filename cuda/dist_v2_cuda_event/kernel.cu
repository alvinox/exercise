#include "hip/hip_runtime.h"
#include "kernel.h"

#define TPB 32

__device__ float distance(float x1, float x2) {
  return sqrt((x2-x1)*(x2-x1));
}

__global__ void distanceKernel(float* d_out, float* d_in, float ref) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const float x = d_in[i];
  d_out[i] = distance(x, ref);
  //printf("i = %2d: dist from %f to %f is %f.\n", i, ref, x, d_out[i]);
}

void distanceArray(float* out, float* in, float ref, int n) {
  hipEvent_t startMemcpy, stopMemcpy;
  hipEvent_t startKernel, stopKernel;

  hipEventCreate(&startMemcpy);
  hipEventCreate(&stopMemcpy);
  hipEventCreate(&startKernel);
  hipEventCreate(&stopKernel);

  float* d_in = NULL;
  float* d_out = NULL;

  hipMalloc(&d_in, n * sizeof(float));
  hipMalloc(&d_out, n * sizeof(float));

  hipEventRecord(startMemcpy);
  for (int i = 0; i < 200; ++i) {
    hipMemcpy(d_in, in, n * sizeof(float), hipMemcpyHostToDevice);
  }
  hipEventRecord(stopMemcpy);

  hipEventRecord(startKernel);
  distanceKernel<<<n/TPB, TPB>>>(d_out, d_in, ref);
  hipEventRecord(stopKernel);

  hipMemcpy(out, d_out, n * sizeof(float), hipMemcpyDeviceToHost);

  hipEventSynchronize(stopMemcpy);
  hipEventSynchronize(stopKernel);

  float memcpyTimeInMs = 0;
  float kernelTimeInMs = 0;
  hipEventElapsedTime(&memcpyTimeInMs, startMemcpy, stopMemcpy);
  hipEventElapsedTime(&kernelTimeInMs, startKernel, stopKernel);
  printf("Data transfer time (ms): %f\n", memcpyTimeInMs);
  printf("Kernel time (ms): %f\n", kernelTimeInMs);

  hipFree(d_in);
  hipFree(d_out);
}
